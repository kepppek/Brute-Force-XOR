#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include ""
#include <conio.h>
#include <string>

using namespace std;

const int lengthText = 11;      // длина текста
const int lengthAlphabet = 27;  // мощность алфавита
const int lengthCombinations = 5;            // количество перебираемых комбинаций
const int lengthKey = 5;           // длина настоящего ключа

const int P = 24576;  // количество потоков

//Функция сложения
__global__ void calculate(unsigned int* hashSource, int* finishThread, unsigned char* intClosed, unsigned char* intRange, unsigned char* alphabet)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= *finishThread)
        return;
    
    unsigned char start[lengthCombinations];
    unsigned char finish[lengthCombinations];

    unsigned char i = 0;

    for (i = 0; i < lengthCombinations; i++)
        start[i] = intRange[id * lengthCombinations + i];

    id = id + 1;

    if (id == P)
        id = id - 1;

    for (i = 0; i < lengthCombinations; i++)
        finish[i] = intRange[id * lengthCombinations + i];


    unsigned int hash = 0;
    unsigned int hashS = *hashSource;
    unsigned char count = 0;
    unsigned char line[lengthCombinations + 1];
    line[lengthCombinations] = '\0';

    while (true)
    {
         //вывод варианта множества на экран
        for (i = 0; i < lengthText; i++)
        {
            // считаем хеш и дешифруем текст
            hash = (hash * 1664525) + ((intClosed[i] - start[i % lengthCombinations] + lengthAlphabet) % lengthAlphabet) + 1013904223;
        }

        // проверка на совпадение хеша
        if (hash == hashS)
        {
             // выводим
            for (i = 0; i < lengthCombinations; i++)
                line[i] = alphabet[start[i]];

            printf("%s = %u = %d \n", line, hash, id);
        }

        hash = 0;

        for (i = 0; i < lengthCombinations; i++)
        {
            // считаем количество совпадений
            if (start[i] == finish[i])
                count++;
            else
                break;
        }


        // условие завершения потока
        if (count == lengthCombinations)
            break;
        else
            count = 0;


        for (i = lengthCombinations - 1; i > -1; i--)
        {
            if (start[i] + 1 == lengthAlphabet)
            {
                start[i] = 0;
            }
            else
            {
                start[i]++;
                break;
            }
        }
    }
}



int main(int argc, char* argv[])
{
    string textSource = "hello world";		  // незашифрованный текст
    string key = "hello";						  // ключ

    unsigned int hash = 0;				      // хеш
    unsigned char intRange[P * lengthCombinations];// границы вычислений для каждого потока
    unsigned char intSource[lengthText];	  // кодированный исходный текст
    unsigned char intOpen[lengthText];		  // кодированный расшифрованный текст
    unsigned char intKey[lengthKey];		  // кодированный ключ
    unsigned char intClosed[lengthText];	  // кодированный зашифрованный текст
    unsigned char alphabet[lengthAlphabet] =  // алфавит
    { 'a', 'b', 'c', 'd' ,'e' ,'f' ,
      'g', 'h' ,'i' ,'j' ,'k' ,'l' ,
      'm' ,'n' ,'o' ,'p' ,'q' ,'r' ,
      's' ,'t' ,'u' ,'v' ,'w' ,'x' ,
      'y' ,'z',' ' };

    // кодируем исходный текст
    for (int i = 0; i < lengthText; i++)
    {
        for (int j = 0; j < lengthAlphabet; j++)
        {
            if (textSource[i] == alphabet[j])
            {
                intSource[i] = (unsigned char)j;
                break;
            }
        }
    }

    // кодируем ключ
    for (int i = 0; i < lengthKey; i++)
    {
        for (int j = 0; j < lengthAlphabet; j++)
        {
            if (key[i] == alphabet[j])
            {
                intKey[i] = (unsigned char)j;
                break;
            }
        }
    }

    //вычисляем хеш 
    for (int i = 0; i < lengthText; i++)
    {
        hash = (hash * 1664525) + intSource[i] + 1013904223;
    }

    // шифруем текст
    for (int i = 0; i < lengthText; i++)
    {
        intClosed[i] = (intSource[i] + intKey[i % lengthKey]) % lengthAlphabet;
    }

    // дешифруем текст
    for (int i = 0; i < lengthText; i++)
    {
        intOpen[i] = (intClosed[i] - intKey[i % lengthKey] + lengthAlphabet) % lengthAlphabet;
    }

    // Определяем границы вычислений для каждого потока
    unsigned long long countCombinations = pow(lengthAlphabet, lengthCombinations);
    unsigned long long rangeCombinations = countCombinations / P + 1; // количество комбинаций на один поток
    unsigned long long number = 0; // номер комбинации с которой начинаем
    int finishThread = 0; // 

    printf("Text = %s\n", textSource);
    printf("Hash of the text = %u\n", hash);
    printf("Key = %s\n", key);
    printf("Number of combinations = %llu\n", countCombinations);
    printf("Number of combinations per thread = %llu\n", rangeCombinations);
    printf("\n===Start calculation===\n\n");


    for (int i = 0; i < P; i++)
    {
        number = i * rangeCombinations;

        if (number >= countCombinations || i == P - 1)
        {
            finishThread = i;

            for (int j = 0; j < lengthCombinations; j++)
                intRange[i * lengthCombinations + (lengthCombinations - j - 1)] = lengthAlphabet - 1;
            
            break;
        }

        for (int j = 0; j < lengthCombinations; j++)
        {
            intRange[i * lengthCombinations + (lengthCombinations - j - 1)] = number % lengthAlphabet;
            number = number / lengthAlphabet;
        }
    }

    unsigned int* Hash;
    int* FinishThread;
    unsigned char* IntClosed;
    unsigned char* IntRange;
    unsigned char* Alphabet;


    clock_t t;
    t = clock();

      //Выделение памяти на устройстве
    hipMalloc((void**)&Hash, sizeof(unsigned int));
    hipMalloc((void**)&FinishThread, sizeof(int));
    hipMalloc((void**)&IntClosed, sizeof(unsigned char) * lengthText);
    hipMalloc((void**)&IntRange, sizeof(unsigned char) * lengthCombinations * P);
    hipMalloc((void**)&Alphabet, sizeof(unsigned char) * lengthAlphabet);

     //Копируем данные на устройство
    hipMemcpy(Hash, &hash, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(FinishThread, &finishThread, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(IntClosed, intClosed, lengthText * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(IntRange, intRange, lengthCombinations * P * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(Alphabet, alphabet, lengthAlphabet * sizeof(unsigned char), hipMemcpyHostToDevice);

    calculate << <96, 256 >> > (Hash, FinishThread, IntClosed, IntRange, Alphabet);

    //Хост ожидает завершения работы девайса
    //hipDeviceSynchronize();

    //Получаем результат
   // hipMemcpy(&b, dev_b, sizeof(int), hipMemcpyHostToHost);

    //Очищаем память на устройстве
    hipFree(Hash);
    hipFree(FinishThread);
    hipFree(IntClosed);
    hipFree(IntRange);
    hipFree(Alphabet);

    t = clock() - t;
    printf("\ntime %.3f\n", ((double)t) / CLOCKS_PER_SEC);


    getch();
    return 0;
}
